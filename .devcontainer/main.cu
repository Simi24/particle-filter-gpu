#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// Funzione di utility per gestire gli errori CUDA
void checkCudaError(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "Errore CUDA: %s - %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// __global__ indica una funzione (kernel) che viene eseguita sulla GPU
__global__ void sum_vectors(float *result, const float *a, const float *b, int n) {
    // Calcola l'indice globale univoco per ogni thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Assicura che il thread non acceda a memoria fuori dai limiti dell'array
    if (idx < n) {
        result[idx] = a[idx] + b[idx];
    }
}

// Funzione main eseguita sulla CPU (host)
int main(void) {
    int N = 1000000; // Numero di elementi nei vettori
    size_t size = N * sizeof(float);

    // 1. Allocazione della memoria sulla CPU (host)
    float *h_a = (float *)malloc(size);
    float *h_b = (float *)malloc(size);
    float *h_c = (float *)malloc(size);

    if (h_a == NULL || h_b == NULL || h_c == NULL) {
        fprintf(stderr, "Errore malloc!\n");
        return 1;
    }

    // Inizializzazione dei dati sulla CPU
    for (int i = 0; i < N; i++) {
        h_a[i] = (float)i;
        h_b[i] = (float)i * 2.0f;
    }

    // 2. Allocazione della memoria sulla GPU (device)
    float *d_a, *d_b, *d_c;
    checkCudaError(hipMalloc(&d_a, size), "hipMalloc d_a");
    checkCudaError(hipMalloc(&d_b, size), "hipMalloc d_b");
    checkCudaError(hipMalloc(&d_c, size), "hipMalloc d_c");

    // 3. Copia dei dati dalla CPU alla GPU
    printf("Copia dei dati da Host a Device...\n");
    checkCudaError(hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice), "hipMemcpy h_a -> d_a");
    checkCudaError(hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice), "hipMemcpy h_b -> d_b");

    // 4. Configurazione ed esecuzione del kernel
    int threadsPerBlock = 256;
    // Calcola il numero di blocchi necessari per coprire tutti gli elementi
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    printf("Lancio del kernel CUDA...\n");
    sum_vectors<<<blocksPerGrid, threadsPerBlock>>>(d_c, d_a, d_b, N);

    // Controlla eventuali errori durante il lancio del kernel
    checkCudaError(hipGetLastError(), "Errore nel lancio del kernel");
    // Sincronizza per essere sicuri che il kernel sia terminato
    checkCudaError(hipDeviceSynchronize(), "hipDeviceSynchronize");

    // 5. Copia del risultato dalla GPU alla CPU
    printf("Copia del risultato da Device a Host...\n");
    checkCudaError(hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost), "hipMemcpy d_c -> h_c");

    // 6. Verifica del risultato
    printf("Verifica del risultato (primi 5 elementi):\n");
    for (int i = 0; i < 5; i++) {
        printf("h_c[%d] = %f (atteso: %f)\n", i, h_c[i], h_a[i] + h_b[i]);
    }

    // 7. Liberazione della memoria
    free(h_a);
    free(h_b);
    free(h_c);
    checkCudaError(hipFree(d_a), "hipFree d_a");
    checkCudaError(hipFree(d_b), "hipFree d_b");
    checkCudaError(hipFree(d_c), "hipFree d_c");

    printf("Operazione completata con successo!\n");

    return 0;
}